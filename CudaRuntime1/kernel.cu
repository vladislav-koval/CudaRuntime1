﻿#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <algorithm>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 1024 // 2^9
#define BLOCKS 32768 // 2^15
#define NUM_VALS THREADS*BLOCKS

using namespace std;

void print_elapsed(clock_t start, clock_t stop) {
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);
}

void array_print(int* arr, int length) {
	int i;
	for (i = 0; i < length; ++i) {
		printf("%1.3f ", arr[i]);
	}
	printf("\n");
}

void array_fill(int* arr, int length) {
	srand(time(NULL));
	int i;
	for (i = 0; i < length; ++i) {
		arr[i] = rand();
	}
}

bool comparison_arrays(int* arr1, int* arr2, int length) {
	for (int i = 0; i < length; i++) {
		if (arr1[i] != arr2[i]) {
			return false;
		}
	}
	return true;
}

int* get_copy_array(int* sourse, int length) {
	int* dest = new int[length];

	for (int i = 0; i < length; i++) {
		dest[i] = sourse[i];
	}
	return dest;
}

int power_ceil(int x) {
	if (x <= 1) return 1;
	int power = 2;
	x--;
	while (x >>= 1) power <<= 1;
	return power;
}

__global__ void bitonic_sort_step(int* dev_values, int j, int k) {
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i ^ j;

	if ((ixj) > i) {
		if ((i & k) == 0) {
			/* Sort ascending */
			if (dev_values[i] > dev_values[ixj]) {
				/* exchange(i,ixj); */
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i & k) != 0) {
			/* Sort descending */
			if (dev_values[i] < dev_values[ixj]) {
				/* exchange(i,ixj); */
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

void bitonic_sort(int* values) {
	int* dev_values;
	size_t size = NUM_VALS * sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(BLOCKS, 1);
	dim3 threads(THREADS, 1);

	int j, k;
	for (k = 2; k <= NUM_VALS; k <<= 1) {
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonic_sort_step <<<blocks, threads>>> (dev_values, j, k);
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}

bool is_bitonic(int*v, int length) {
	bool was_decreasing = v[length - 1] > v[0];
	int num_inflections = 0;
	for (int i = 0; i < length && num_inflections <= 2; i++) {
		bool is_decreasing = v[i] > v[(i + 1) % length];
		// Check if this element and next one are an inflection.
		if (was_decreasing != is_decreasing) {
			num_inflections++;
			was_decreasing = is_decreasing;
		}
	}

	return 2 == num_inflections;
}

int main(void)
{
	clock_t start, stop;

	int length = 0;
	cout << "Enter length of the array: ";
	cin >> length;

	int* values = (int*)malloc(NUM_VALS * sizeof(int));
	array_fill(values, NUM_VALS);
	int* temp = get_copy_array(values, NUM_VALS);


	sort(temp, temp + NUM_VALS);


	start = clock();
	bitonic_sort(values);
	stop = clock();

	cout << "is_bitonic: " << is_bitonic(values, NUM_VALS) << endl;;
	cout << "is equals: " << comparison_arrays(values, temp, NUM_VALS) << endl;
	print_elapsed(start, stop);
}
